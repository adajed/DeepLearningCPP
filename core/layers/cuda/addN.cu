#include "hip/hip_runtime.h"
#include "layers/addN.h"

namespace graphdl
{
namespace core
{
namespace layers
{
namespace cuda
{
namespace
{
__global__ void addNKernel(int n, size_t size, float** xs, float* y)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
    {
        y[id] = 0;
        for (int i = 0; i < n; ++i) y[id] += xs[i][id];
    }
}

__global__ void addNGradientKernel(int n, size_t size, const float* yG,
                                   float** xGs)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
    {
        for (int i = 0; i < n; ++i) xGs[i][id] = yG[id];
    }
}

}  // namespace

void runAddNDevice(int n, size_t size, float** xs, float* y)
{
    const int BLOCK_SIZE = 256;
    const int NUM_BLOCKS = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    float** xsDevice;
    hipMalloc((void**)&xsDevice, n * sizeof(float*));
    hipMemcpy(xsDevice, xs, n * sizeof(float*), hipMemcpyHostToDevice);
    addNKernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(n, size, xsDevice, y);
    hipDeviceSynchronize();
    hipFree(xsDevice);
}

void runAddNGradientDevice(int n, size_t size, float* yGrad, float** xGrads)
{
    const int BLOCK_SIZE = 256;
    const int NUM_BLOCKS = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    float** xGradsDevice;
    hipMalloc((void**)&xGradsDevice, n * sizeof(float*));
    hipMemcpy(xGradsDevice, xGrads, n * sizeof(float*),
               hipMemcpyHostToDevice);
    addNGradientKernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(n, size, yGrad,
                                                   xGradsDevice);
    hipDeviceSynchronize();
    hipFree(xGradsDevice);
}

}  // namespace cuda
}  // namespace layers
}  // namespace core
}  // namespace graphdl
