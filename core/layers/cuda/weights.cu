#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "weights.h"

namespace graphdl
{
namespace core
{
namespace cuda
{
__global__ void setup_kernel(hiprandState* state) {}

__global__ void initWeightsKernel(hiprandState* cuState, size_t N, float* output)
{
}

extern "C" void initWeights(float* memory, size_t N) {}

}  // namespace cuda
}  // namespace core
}  // namespace graphdl
