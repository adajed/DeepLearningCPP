#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "weights.h"

namespace graphdl
{
namespace core
{
namespace cuda
{
__global__ void setup_kernel(hiprandState* state)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void initWeightsKernel(hiprandState* state, size_t size,
                                  float* output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) output[id] = hiprand_uniform(state + id) * 2. - 1.;
}

extern "C" void initWeights(float* memory, size_t size)
{
    const int BLOCK_SIZE = 256;
    const int NUM_BLOCKS = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hiprandState* states;
    hipMalloc(&states, size * sizeof(hiprandState));
    setup_kernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(states);
    initWeightsKernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(states, size, memory);
    hipDeviceSynchronize();
}

}  // namespace cuda
}  // namespace core
}  // namespace graphdl
