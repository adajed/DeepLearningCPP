#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

namespace graphdl
{
namespace core
{
namespace cuda
{
__global__ void setupKernel(hiprandState* state, size_t seed)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void uniformRandomKernel(hiprandState* state, float* memory,
                                    size_t size, float min, float max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) memory[id] = hiprand_uniform(state + id) * (max - min) + min;
}

__global__ void normalRandomKernel(hiprandState* state, float* memory,
                                   size_t size, float mean, float stddev)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) memory[id] = stddev * hiprand_normal(state + id) + mean;
}

void uniformRandom(float* memory, size_t size, float min, float max,
                   size_t seed)
{
    const int BLOCK_SIZE = 256;
    const int NUM_BLOCKS = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hiprandState* states;
    hipMalloc(&states, size * sizeof(hiprandState));
    setupKernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(states, seed);
    uniformRandomKernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(states, memory, size, min,
                                                    max);
    hipDeviceSynchronize();
}

void normalRandom(float* memory, size_t size, float mean, float stddev,
                  size_t seed)
{
    const int BLOCK_SIZE = 256;
    const int NUM_BLOCKS = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hiprandState* states;
    hipMalloc(&states, size * sizeof(hiprandState));
    setupKernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(states, seed);
    normalRandomKernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(states, memory, size, mean,
                                                   stddev);
    hipDeviceSynchronize();
}

}  // namespace cuda
}  // namespace core
}  // namespace graphdl
