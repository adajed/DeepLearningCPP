#include "hip/hip_runtime.h"
#include "layers/matmul.h"

namespace graphdl
{
namespace core
{
namespace layers
{
namespace cuda
{
template <int TILE_SIZE, bool tran1, bool tran2>
__global__ void matmulKernel(int n, int m, int k, float* X1, float* X2,
                             float* Y)
{
    __shared__ float tile_X1[TILE_SIZE * TILE_SIZE];
    __shared__ float tile_X2[TILE_SIZE * TILE_SIZE];

    int pos = TILE_SIZE * threadIdx.x + threadIdx.y;
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;
    float tmp = 0.;

    for (int t = 0; t < m; t += TILE_SIZE)
    {
        if (t + threadIdx.y < m)
        {
            if (tran1)
                tile_X1[pos] = X1[n * (t + threadIdx.y) + row];
            else
                tile_X1[pos] = X1[m * row + t + threadIdx.y];
        }
        else
            tile_X1[pos] = 0.;

        if (t + threadIdx.x < m)
        {
            if (tran2)
                tile_X2[pos] = X2[m * col + t + threadIdx.x];
            else
                tile_X2[pos] = X2[k * (t + threadIdx.x) + col];
        }
        else
            tile_X2[pos] = 0.;

        __syncthreads();

        for (int i = 0; i < TILE_SIZE; ++i)
            tmp += tile_X1[TILE_SIZE * threadIdx.x + i] *
                   tile_X2[TILE_SIZE * i + threadIdx.y];

        __syncthreads();
    }

    if (row < n && col < k) Y[k * row + col] = tmp;
}

__global__ void matmulGradientKernel(int n, int m, int k, float* X1, float* X2,
                                     float* Ygrad, float* X1grad, float* X2grad)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n * m)
    {
        int x = id / m;
        int y = id % m;
        X1grad[id] = 0.;
        for (int i = 0; i < k; ++i)
            X1grad[id] += X2[k * y + i] * Ygrad[k * x + i];
    }
    else
    {
        id -= n * m;
        if (id < m * k)
        {
            int x = id / k;
            int y = id % k;
            X2grad[id] = 0.;
            for (int i = 0; i < n; ++i)
                X2grad[id] += X1[m * i + x] * Ygrad[k * i + y];
        }
    }
}

extern "C" void runMatmulDevice(int n, int m, int k, float* X1, float* X2,
                                float* Y)
{
    const int TILE_SIZE = 16;
    dim3 GRID((n + TILE_SIZE - 1) / TILE_SIZE, (k + TILE_SIZE - 1) / TILE_SIZE);
    dim3 BLOCK(TILE_SIZE, TILE_SIZE);
    matmulKernel<TILE_SIZE, false, false><<<GRID, BLOCK>>>(n, m, k, X1, X2, Y);
}

extern "C" void runMatmulGradientDevice(int n, int m, int k, float* X1,
                                        float* X2, float* Ygrad, float* X1grad,
                                        float* X2grad)
{
    const int TILE_SIZE = 16;
    dim3 BLOCK(TILE_SIZE, TILE_SIZE);
    dim3 GRID1((n + TILE_SIZE - 1) / TILE_SIZE,
               (m + TILE_SIZE - 1) / TILE_SIZE);
    dim3 GRID2((m + TILE_SIZE - 1) / TILE_SIZE,
               (k + TILE_SIZE - 1) / TILE_SIZE);

    matmulKernel<TILE_SIZE, false, true>
        <<<GRID1, BLOCK>>>(n, k, m, Ygrad, X2, X1grad);
    matmulKernel<TILE_SIZE, true, false>
        <<<GRID2, BLOCK>>>(m, n, k, X1, Ygrad, X2grad);
}

}  // namespace cuda
}  // namespace layers
}  // namespace core
}  // namespace graphdl
