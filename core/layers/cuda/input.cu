#include <hip/hip_runtime.h>
#include "input.h"

namespace graphdl
{
namespace core
{
namespace cuda
{
extern "C" void copyInput(float* dest, float* src, size_t N)
{
    hipMemcpy(dest, src, N * sizeof(float), hipMemcpyHostToDevice);
}

}  // namespace cuda
}  // namespace core
}  // namespace graphdl
