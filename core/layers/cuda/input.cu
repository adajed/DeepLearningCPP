#include "input.h"

#include <hip/hip_runtime.h>

namespace graphdl
{
namespace core
{
namespace cuda
{
extern "C" void copyInput(float* dest, float* src, size_t N)
{
    hipMemcpy(dest, src, N * sizeof(float), hipMemcpyHostToDevice);
}

}  // namespace cuda
}  // namespace core
}  // namespace graphdl
