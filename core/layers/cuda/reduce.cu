#include "hip/hip_runtime.h"
#include "layers/reduce.h"
#include "reduceUtils.h"

namespace graphdl
{
namespace core
{
namespace layers
{
namespace cuda
{
namespace
{
__global__ void reduceSumGradientKernel(const float* yGrad, float* xGrad,
                                        size_t size, size_t reduceSize)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) xGrad[id] = yGrad[id / reduceSize];
}

__global__ void reduceSumFrontKernel(const float* x, float* y,
                                     size_t reduceSize, size_t outSize)
{
    const int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < outSize)
    {
        float s = 0.;
        for (int i = 0; i < reduceSize; ++i) s += x[i * outSize + id];

        y[id] = s;
    }
}

__global__ void reduceSumFrontGradientKernel(const float* yGrad, float* xGrad,
                                             size_t size, size_t outSize)
{
    const int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) xGrad[id] = yGrad[id % outSize];
}

}  // namespace

void runReduceBackDevice(const float* x, float* y, size_t outSize,
                         size_t reduceSize, ReduceType /*reduceType*/)
{
    reduce<ReduceOpCuda::kSUM>(x, y, outSize, reduceSize);
}

void runReduceBackGradientDevice(const float* x, const float* y,
                                 const float* yGrad, float* xGrad,
                                 size_t outSize, size_t reduceSize,
                                 ReduceType /*reduceType*/)
{
    const int BLOCK_SIZE = 256;
    const int NUM_BLOCKS = (outSize * reduceSize + BLOCK_SIZE - 1) / BLOCK_SIZE;

    reduceSumGradientKernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(
        yGrad, xGrad, outSize * reduceSize, reduceSize);
}

void runReduceFrontDevice(const float* x, float* y, size_t outSize,
                          size_t reduceSize, ReduceType /*reduceType*/)
{
    const int BLOCK_SIZE = 256;
    const int NUM_BLOCKS = (outSize + BLOCK_SIZE - 1) / BLOCK_SIZE;

    reduceSumFrontKernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(x, y, reduceSize, outSize);
}

void runReduceFrontGradientDevice(const float* x, const float* y,
                                  const float* yGrad, float* xGrad,
                                  size_t outSize, size_t reduceSize,
                                  ReduceType /*reduceType*/)
{
    const int BLOCK_SIZE = 256;
    const int NUM_BLOCKS = (outSize * reduceSize + BLOCK_SIZE - 1) / BLOCK_SIZE;

    reduceSumFrontGradientKernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(
        yGrad, xGrad, outSize * reduceSize, outSize);
}

}  // namespace cuda
}  // namespace layers
}  // namespace core
}  // namespace graphdl
