#include "hip/hip_runtime.h"
#include "layers/cuda/macros.h"
#include "layers/dataFormatReshape.h"

namespace graphdl
{
namespace core
{
namespace layers
{
namespace cuda
{
namespace
{
__global__ void nhwc2nchw_kernel(const float* in, float* out, int N, int C,
                                 int X, int Y)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int n = blockIdx.z * blockDim.z + threadIdx.z;
    int c = n % C;
    n /= C;

    if (n < N && x < X && y < Y)
        out[((n * C + c) * X + x) * Y + y] = in[((n * X + x) * Y + y) * C + c];
}

__global__ void nchw2nhwc_kernel(const float* in, float* out, int N, int X,
                                 int Y, int C)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int n = blockIdx.z * blockDim.z + threadIdx.z;
    int c = n % C;
    n /= C;

    if (n < N && x < X && y < Y)
        out[((n * X + x) * Y + y) * C + c] = in[((n * C + c) * X + x) * Y + y];
}

}  // namespace

void runNhwc2NchwDevice(const float* in, float* out, int* outShape)
{
    const int TILE = 8;
    const dim3 BLOCK(TILE, TILE, TILE);
    const dim3 GRID((outShape[2] + TILE - 1) / TILE,
                    (outShape[3] + TILE - 1) / TILE,
                    (outShape[0] * outShape[1] + TILE - 1) / TILE);

    nhwc2nchw_kernel<<<GRID, BLOCK>>>(in, out, outShape[0], outShape[1],
                                      outShape[2], outShape[3]);
}

void runNchw2NhwcDevice(const float* in, float* out, int* outShape)
{
    const int TILE = 8;
    const dim3 BLOCK(TILE, TILE, TILE);
    const dim3 GRID((outShape[1] + TILE - 1) / TILE,
                    (outShape[2] + TILE - 1) / TILE,
                    (outShape[0] * outShape[3] + TILE - 1) / TILE);

    nchw2nhwc_kernel<<<GRID, BLOCK>>>(in, out, outShape[0], outShape[1],
                                      outShape[2], outShape[3]);
}

}  // namespace cuda
}  // namespace layers
}  // namespace core
}  // namespace graphdl
